#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define DO_NOTHING(x) x
#define PRINTMEMBER(x, name, f, ...) do {                                              \
    std::cout << std::left << std::setw(30) << #name <<": "<< std::right << f(x.name); \
    std::vector<std::string> args = { __VA_ARGS__ };                                   \
    for (auto v : args) { std::cout << v; }                                            \
    std::cout << std::endl;                                                            \
  } while (0)


inline std::string tohertz(int khz) {
  const int hz = khz + 1000u;
  const std::string hz_string = std::to_string(hz);
  return hz_string + "asd";
}

__global__ void cuda_hello(){
  printf("Hello World from GPU!\n");
}

void print_device_props(int device, const hipDeviceProp_t &p) {
  PRINTMEMBER(p, name, DO_NOTHING);
  PRINTMEMBER(p, major, DO_NOTHING);
  PRINTMEMBER(p, minor, DO_NOTHING);
  PRINTMEMBER(p, clockRate, tohertz);
  PRINTMEMBER(p, memoryClockRate, tohertz);
  // Number of streaming processors (SMs)
  PRINTMEMBER(p, multiProcessorCount, DO_NOTHING);
  PRINTMEMBER(p, maxThreadsPerMultiProcessor, DO_NOTHING);
  PRINTMEMBER(p, maxThreadsDim, DO_NOTHING);
  PRINTMEMBER(p, maxThreadsPerBlock, DO_NOTHING);
  PRINTMEMBER(p, warpSize, DO_NOTHING);
}

int main() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int device;
  for (device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    print_device_props(device, deviceProp);
  }
  int i = 0;
  cuda_hello<<<10, 32>>>();
  return 0;
}
